#include "hip/hip_runtime.h"
#include "include/bellman-ford.h"
#include <cstdio>

void unroll(int** matrix, int* vector, int N){
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            hipMemcpy(&vector[(i * N)], matrix[i], sizeof(int) * N, hipMemcpyHostToDevice);
        }
    }
}

__global__ void initialize(int* distance, int* predecessor, int INFINITE, int source){
    int i = blockIdx.x;
    if (i != source)
        distance[i] = INFINITE;
    else
        distance[i] = 0;
    predecessor[i] = -1;
}

__global__ void bf_iter_nodes(int* edges, int* distance, int* predecessor, int* changes, int n){

    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int increment = blockDim.x * gridDim.x;
    if(global_tid >= n)
        return;
    for(int i = 0; i < n; i++){
        for(int j = global_tid; j < n; j+= increment){
            int edge = edges[(i * n) + j];
            int change_distance = (edge != 0 && distance[i] + edge < distance[j]);
            distance[j] = ((distance[i] + edge) * change_distance) + (distance[j] * !change_distance);
            changes[j] += change_distance;
            predecessor[j] = (i * change_distance) + (predecessor[j] * !change_distance);
        }
    }
}

void find_distances_nodes(int* edges, int* distance, int* predecessor, int n){
    int* d_changes;

    unsigned int size = sizeof(int) * n;
    hipMalloc((void**) &d_changes, size);
    int *total_changes = (int*)malloc(size);

    for(int steps = 0; steps < n - 1; steps ++){

        hipMemset(d_changes, 0, size);
        bf_iter_nodes<<<512, 1024>>>(edges, distance, predecessor, d_changes, n);
        Max_Sequential_Addressing_Shared<<<512, 1024, 1024 * sizeof(int)>>>(d_changes, n);
        hipMemcpy(total_changes, d_changes, size, hipMemcpyDeviceToHost);

        if(total_changes[0] == 0)
            break;
    }

    free(total_changes);
    hipFree(d_changes);
}

__global__ void bf_iter_edge(edge_array* edges, int* distance, int* predecessor, int* changes, int n){

    int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    int increment = blockDim.x * gridDim.x;
    for(int i = 0; i < n; i++){
        if(global_tid < *edges[i].size){
            // printf("(%d, %d)", *edges[i].size, global_tid);
            for(int idx = global_tid; idx < *edges[i].size; idx+= increment){
                edge edge = edges[i].values[idx];
                int change_distance = distance[edge.source] + edge.weight < distance[edge.destination];
                if(distance[edge.source] + edge.weight < distance[edge.destination])
                    changes[edge.destination] += 1;
                // printf("(%d, %d, %d) %d\n", edge.source, edge.destination, edge.weight, n);
                // distance[edge.destination] = ((distance[edge.source] + edge.weight) * change_distance) + (distance[edge.destination] * !change_distance);
                // changes[edge.destination] += change_distance;
                // printf("%d,%d\n", edges[i].values[idx].source,edges[i].values[idx].destination);
                // predecessor[edge.destination] = (edge.source * change_distance) + (predecessor[edge.destination] * !change_distance);
            }
        }
    }
}

void find_distances_edges(edge_array* edges, int* distance, int* predecessor, int n){

    int* d_changes;
    unsigned int size = sizeof(int) * n;

    int* total_changes = (int*)malloc(size);

    hipMalloc((void**) &d_changes, size);

    for(int steps = 0; steps < n - 1; steps ++){
        hipMemset(d_changes, 0, size);
        bf_iter_edge<<<n,n>>>(edges, distance, predecessor, d_changes, n);
        printf("%s\n", hipGetErrorString(hipGetLastError()));
        // Max_Sequential_Addressing_Shared<<<512,  1024, 1024 * sizeof(int)>>>(d_changes, n);
        hipMemcpy(total_changes, d_changes, size, hipMemcpyDeviceToHost);
        if(total_changes[0] == 0)
            break;
    }
    free(total_changes);
    hipFree(d_changes);
}

__global__ void check_negative_cycle(int* edges, int source, int* distance, int* negative_cycles, int N){
    int i = blockIdx.x;
    int j = threadIdx.x;
    if((i * N) + j >= N || j >= N || i >= N)
        return;
    int edge = edges[(i * N) + j];
    negative_cycles[j] = edge != 0 && i != source && distance[i] + edge < distance[j];
}

int find_negative_cycles_nodes(int* edges, int* distance, int source, unsigned int N){
    int *d_negative_cycles;

    hipMalloc(&d_negative_cycles, sizeof(int) * N);

    hipMemset(d_negative_cycles, 0, sizeof(int) * N);

    check_negative_cycle<<<512,1024>>>(edges, source, distance, d_negative_cycles, N);
    int negative_cycle = 0;

    Max_Sequential_Addressing_Shared<<<512,1024, 1024 * sizeof(int)>>>(d_negative_cycles, N);
    hipMemcpy(&negative_cycle, d_negative_cycles, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_negative_cycles);
    return negative_cycle;
}

__global__ void internal_neg_cycles_edges(edge_array* edges, int* distance, int* negative_cycles, int N){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i >= N)
        return;
    // edge edge = edges[i];
    // negative_cycles[edge.destination] = distance[edge.source] + edge.weight < distance[edge.destination];
}

int find_negative_cycles_edges(edge_array* edges, int* distance, int N, int size_edges){
    int *d_negative_cycles;
    int* negative_cycle = (int*)malloc(sizeof(int) * N);
    memset(negative_cycle, 0, sizeof(int) * N);

    hipMalloc(&d_negative_cycles, sizeof(int) * N);

    hipMemset(d_negative_cycles, 0, sizeof(int) * N);

    internal_neg_cycles_edges<<<512,1024>>>(edges, distance, d_negative_cycles, size_edges);
    Max_Sequential_Addressing_Shared<<<512,1024, 1024 * sizeof(int)>>>(d_negative_cycles, N);
    hipMemcpy(negative_cycle, d_negative_cycles, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_negative_cycles);
    int neg = negative_cycle[0];
    free(negative_cycle);
    return neg;
}


bellman_ford_return* find_distances_iterate_over_nodes(graph* graph, int source){
    double t_start;

    const unsigned int N = graph->nodes.size;
    int* distance = (int*)malloc(sizeof(int)* N);
    int* predecessor = (int*)malloc(sizeof(int)* N);

    int* d_distance;
    int* d_predecessor;
    int* d_edges;

    hipMalloc((void**)&d_distance, sizeof(int) * N);
    hipMalloc((void**)&d_predecessor, sizeof(int) * N);
    hipMalloc((void**)&d_edges, sizeof(int) * graph->nodes.size * graph->nodes.size);

    unroll(graph->edges.values, d_edges, graph->nodes.size);

    t_start = omp_get_wtime();
    const int INFINITE = find_infinite(d_edges, N * N);
    float inf_time = omp_get_wtime() - t_start;

    t_start = omp_get_wtime();
    initialize<<<N,1>>>(d_distance, d_predecessor, INFINITE, 0);
    float init_time = omp_get_wtime() - t_start;

    hipMemcpy(distance, d_distance, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(predecessor, d_predecessor, sizeof(int) * N, hipMemcpyDeviceToHost);

    t_start = omp_get_wtime();
    find_distances_nodes(d_edges, d_distance, d_predecessor, N);
    float rel_time = omp_get_wtime() - t_start;

    t_start = omp_get_wtime();
    int negative_cycles = find_negative_cycles_nodes(d_edges, d_distance, source, N);
    float neg_time = omp_get_wtime() - t_start;

    bellman_ford_return* return_value = (bellman_ford_return*)malloc(sizeof(bellman_ford_return));

    hipMemcpy(distance, d_distance, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(predecessor, d_predecessor, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(d_distance);
    hipFree(d_predecessor);
    hipFree(d_edges);

    int_array distances;
    distances.size = graph->nodes.size;
    distances.values = distance;

    int_array predecessors;
    predecessors.size = graph->nodes.size;
    predecessors.values = predecessor;

    return_value->distances = distances;
    return_value->predecessors = predecessors;
    return_value->negative_cycles = mmin(1, 0);
    return_value->infinite_time = inf_time;
    return_value->init_time = init_time;
    return_value->relaxation_time = rel_time;
    return_value->negative_cycle_time = neg_time;

    return  return_value;
}

edge_array* get_edges(graph* graph){

    int* n_edges = (int*)malloc(sizeof(int) * graph->nodes.size);
    memset(n_edges, 0, sizeof(int) * graph->nodes.size);

    for(int i = 0; i < graph->nodes.size; i++){
        for(int j = 0; j < graph->nodes.size; j++){
            n_edges[i] += graph->edges.values[i][j] != 0;
        }
    }
    
    int* current_edges = (int*)malloc(sizeof(int) * graph->nodes.size);
    memset(current_edges, 0, sizeof(int) * graph->nodes.size);

    edge_array* edges = (edge_array*)malloc(sizeof(edge_array) * graph->nodes.size);

    for(int i = 0; i < graph->nodes.size; i++){
        edges[i].values = (edge*)malloc(sizeof(edge) * n_edges[i]);
        edges[i].size = (int*)malloc(sizeof(int));
        *edges[i].size = n_edges[i];
        for(int j = 0; j < graph->nodes.size; j++){
            if(graph->edges.values[i][j] != 0){
                edges[i].values[current_edges[i]].source = i;
                edges[i].values[current_edges[i]].destination = j;
                edges[i].values[current_edges[i]].weight = graph->edges.values[i][j];
                current_edges[i] ++;
            }
        }
    }
    free(n_edges);
    free(current_edges);
    
    return edges;
}

bellman_ford_return* find_distances_iterate_over_edges(graph* graph, int source){
    double t_start;

    int N = graph->nodes.size;
    int* distance = (int*)malloc(sizeof(int)* N);
    int* predecessor = (int*)malloc(sizeof(int)* N);

    int* d_distance;
    int* d_predecessor;
    int* d_edges;
    edge_array* d_edge_array;

    hipMalloc((void**)&d_distance, sizeof(int) * N);
    hipMalloc((void**)&d_predecessor, sizeof(int) * N);
    hipMalloc((void**)&d_edges, sizeof(int) * graph->nodes.size * graph->nodes.size);

    unroll(graph->edges.values, d_edges, graph->nodes.size);

    edge_array* edges_array = get_edges(graph);
    hipMalloc(&d_edge_array, sizeof(edge_array*) * N);
    edge_array host_array[N];
    for(int i = 0; i < N; i++){
        hipMalloc(&host_array[i].values, sizeof(edge) * *edges_array[i].size);
        hipMalloc(&host_array[i].size, sizeof(int));
        hipMemcpy(host_array[i].values, edges_array[i].values, sizeof(edge) * *edges_array[i].size, hipMemcpyHostToDevice); 
        hipMemcpy(host_array[i].size, edges_array[i].size, sizeof(int), hipMemcpyHostToDevice); 
    }

    hipMemcpy(d_edge_array, host_array, sizeof(edge_array*) * N, hipMemcpyHostToDevice);

    t_start = omp_get_wtime();
    const int INFINITE = find_infinite(d_edges, N * N);
    float inf_time = omp_get_wtime() - t_start;
    t_start = omp_get_wtime();
    initialize<<<N,1>>>(d_distance, d_predecessor, INFINITE, 0);
    float init_time = omp_get_wtime() - t_start;

    hipFree(d_edges);
    
    t_start = omp_get_wtime();
    find_distances_edges(d_edge_array, d_distance, d_predecessor, N);
    float rel_time = omp_get_wtime() - t_start;

    t_start = omp_get_wtime();
    // int negative_cycles = find_negative_cycles_edges(d_edge_array, d_distance, N, edges_array->size);
    float neg_time = omp_get_wtime() - t_start;

    bellman_ford_return* return_value = (bellman_ford_return*)malloc(sizeof(bellman_ford_return));

    memset(distance, 0, sizeof(int) * N);
    memset(predecessor, 0, sizeof(int) * N);
    // hipMemcpy(distance, d_distance, sizeof(int) * N, hipMemcpyDeviceToHost);
    // hipMemcpy(predecessor, d_predecessor, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(d_distance);
    hipFree(d_predecessor);
    hipFree(d_edge_array);
    for(int i = 0; i < N; i++){
        hipFree(host_array[i].values);
        hipFree(host_array[i].size);
    }
    printf("\n%s\n", hipGetErrorString(hipGetLastError()));

    int_array distances;
    distances.size = N;
    distances.values = distance;

    int_array predecessors;
    predecessors.size = N;
    predecessors.values = predecessor;

    return_value->distances = distances;
    return_value->predecessors = predecessors;
    // return_value->negative_cycles = mmin(1, negative_cycles);
    return_value->init_time = init_time;
    return_value->infinite_time = inf_time;
    return_value->relaxation_time = rel_time;
    return_value->negative_cycle_time = neg_time;

    return  return_value;
}
